#include "hip/hip_runtime.h"
#include "LumenOpus/hittable/sphere.h"
#include ""

namespace LumenOpus
{
	Spheres::Spheres(uint64_t maxSize)
		: m_maxArraySize(maxSize)
		, m_dataSize(maxSize * ARRAY_COUNT)
		, ArraySize(0)
	{
		AllocateData();
	}

	Spheres::~Spheres()
	{
		FreeData();
	}

	__host__ __device__ void Spheres::Add(
		float x, 
		float y, 
		float z,
		float radius,
		float ka,
		float kd,
		float ks,
		float shininess,
		float r,
		float g,
		float b,
		float a
	)
	{
		// Casual return cause idk what happens on that gpu
		if (ArraySize == m_maxArraySize) return;

		float* ptr = &Data[ArraySize];
		*ptr = x;
		
		ptr += m_maxArraySize;
		*ptr = y;

		ptr += m_maxArraySize;
		*ptr = z;
		
		ptr += m_maxArraySize;
		*ptr = radius;
		
		ptr += m_maxArraySize;
		*ptr = ka;
		
		ptr += m_maxArraySize;
		*ptr = kd;
		
		ptr += m_maxArraySize;
		*ptr = ks;
		
		ptr += m_maxArraySize;
		*ptr = shininess;
		
		ptr += m_maxArraySize;
		*ptr = r;
		
		ptr += m_maxArraySize;
		*ptr = g;
		
		ptr += m_maxArraySize;
		*ptr = b;
		
		ptr += m_maxArraySize;
		*ptr = a;
		
		ArraySize++;
	}

	__host__ __device__ void Spheres::AllocateData()
	{
		Data = new float[m_dataSize];
	}

	__host__ __device__ void Spheres::FreeData()
	{
		delete[] Data;
		m_dataSize = m_maxArraySize = ArraySize = 0;
		Data = nullptr;
	}

	__host__ Spheres** Spheres::MakeItDevice(const Spheres& host)
	{
		Spheres** d_s;
		checkCudaErrors(hipMalloc((void**)&d_s, sizeof(Spheres*)));

		CreateSpheresDevice<<<1, 1>>>(d_s, host.m_maxArraySize);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		CopyToDevice(d_s, host);

		return d_s;
	}
	__host__ void Spheres::DeleteDevice(Spheres** spheres)
	{
		DeleteSphereDevice<<<1, 1>>>(spheres);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipFree(spheres));
	}

	__host__ void Spheres::CopyToDevice(Spheres** d_s, const Spheres& h_s)
	{
		int gridSize = 0, blockSize = 0;
		checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, CopySphereToDevice, 0));

		float *d_host;

		size_t copySize = h_s.m_dataSize * sizeof(float);
		checkCudaErrors(hipMalloc(&d_host, copySize));
		checkCudaErrors(hipMemcpy(d_host, h_s.Data, copySize, hipMemcpyDefault));

		CopySphereToDevice<<<gridSize, blockSize>>>(d_s, d_host, h_s.m_dataSize, h_s.ArraySize);

		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipFree(d_host));
	}

	__global__ void CreateSpheresDevice(Spheres** s, size_t maxSize)
	{
		if (threadIdx.x | blockIdx.x) return;
		(*s) = new Spheres(maxSize);
#ifndef NDEBUG
		(*s)->m_type = DeviceType::GPU;
#endif
	}

	__global__ void DeleteSphereDevice(Spheres** s)
	{
		if (threadIdx.x | blockIdx.x) return;
		delete* (s);
	}

	__global__ void CopySphereToDevice(
		Spheres** s,
		const float* host,
		int N,
		uint64_t Size
		)
	{
		float* out = (*s)->Data;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
			out[i] = host[i];

		if (threadIdx.x | blockIdx.x) return;

		(*s)->ArraySize = Size;
	}
}
