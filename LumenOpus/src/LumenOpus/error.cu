#include "LumenOpus/error.h"

void LumenOpus::check_cuda(hipError_t result, const char* const func, const char* const file, const int line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}
