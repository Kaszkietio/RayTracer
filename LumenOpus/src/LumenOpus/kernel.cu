#include "hip/hip_runtime.h"
﻿#include "LumenOpus/kernel.h"
#include "LumenOpus/utils.h"

#include ""
#include "hip/hip_vector_types.h"

#include <stdio.h>
#include <cstdint>

__global__ void LumenOpus::render_pixel(
    uint32_t* data, 
	Spheres** spheres,
	Camera camera,
    float4* d_rayOrigin, 
    float angleYAxis,
    int32_t max_x, 
    int32_t max_y)
{
	constexpr float piRatio = 3.14159265358979323846f / 180.0f;
	float imageAspectRatio = float(max_x) / float(max_y);

	Spheres* hittable = *spheres;

	float4 rayOrigin = make_float4(camera.Position);
	rayOrigin.w = 1.0f;

	// Rotation data
	float radians = piRatio * angleYAxis;
	float cosine = cosf(radians);
	float sine = sinf(radians);

	// Fun time
	float4 spherePosition = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	float4 lightPosition = make_float4(0.0f, 2.0f, 1.0f, 1.0f);

	uint32_t backColor = to_rgba(0.0f, 0.0f, 0.0f, 1.0f);
	float closestT;

    uint32_t x, y;
    float4 rayDirection;
	bool isHit;
	HitRecord rec;

    float4 mat{};
	float4 objectColor{};
    float4 hitPoint{};
	float4 lightColor = make_float4(1.0f);


    for (uint32_t index = threadIdx.x + blockDim.x * blockIdx.x; index < max_x * max_y; index += blockDim.x * gridDim.x)
    {
        x = index % max_x;
        y = index / max_x;

        rayDirection = make_float4(
            (2.0f * float(x) / float(max_x) - 1.0f) * imageAspectRatio,
            2.0f * float(y) / float(max_y) - 1.0f,
            -1.0f,
            0.0f
        );

        rayDirection = camera.GetDirection(float(x), float(y));

        Ray ray(rayOrigin, rayDirection);
        isHit = HitSpheres(hittable, &ray, 0.00f, INFINITY, &rec);
        closestT = rec.t;

        if (!isHit || closestT < 0)
        {
            data[index] = backColor;
            continue;
        }

        hitPoint = ray.at(rec.t);
        spherePosition = hittable->GetPosition(rec.sphereId);
        mat = hittable->GetMat(rec.sphereId);
        objectColor = hittable->GetColor(rec.sphereId);

		float4 result = PhongModel(
            mat,
            rayOrigin,
            spherePosition,
            lightPosition,
            lightColor,
            hitPoint,
            objectColor
        );

        data[index] = to_rgba(result);
    }
}

__host__ __device__ LumenOpus::SphereHit LumenOpus::is_sphere_hit(const float3& rayOrigin, const float3& spherePosition, const float& sphereRadius)
{
    return {};
}

__host__ __device__ bool LumenOpus::is_sphere_hit(
    const float4& rayOrigin, 
	const float4& rayDirection,
	const float4& spherePosition, 
    const float& sphereRadius, 
    float& closestT)
{
    float4 origin = rayOrigin - spherePosition;

    // (bx^2 + by^2 + bz^2)t^2 + 2(axbx + ayby + azbz)t + (ax^2 + ay^2 + az^2 - r^2)
    // a - ray origin
    // b - ray direction
    // r - radius
    // t - hit distance 
    float a = dot(rayDirection, rayDirection);
    float b = 2 * dot(origin, rayDirection);
    float c = dot(origin, origin) - sphereRadius * sphereRadius;

	float delta = b * b - 4.0f * a * c;

    // return background color if no hit
    if (delta < 0.0f) return false;

    closestT = (-b - sqrtf(delta)) / (2.0f * a);
    return true;
}

__host__ __device__ float4 LumenOpus::PhongModel(
    const float4& mat,
    const float4& rayOrigin,        // has to have 1 in w coord
    const float4& spherePosition,   // has to have 1 in w coord
    const float4& lightPosition,    // has to have 1 in w coord
    const float4& lightColor,
    const float4& hitPoint,         // has to have 1 in w coord
    const float4& objectColor
)
{
    float4 normal = normalize(hitPoint - spherePosition);
    float4 viewDir = normalize(rayOrigin - spherePosition);
    float4 lightDirection = normalize(lightPosition - hitPoint);
    float4 reflectDir = LumenOpus::reflect(-lightDirection, normal);

    const float& ambientStrength = mat.x;
    const float& diffuseStrenght = mat.y;
    const float& specularStrength = mat.z;
    const float& shininess = mat.w;

    // TODO: Uzaleznic diffuse i specular od odleglosci
    float4 ambient = ambientStrength * lightColor;
    float4 diffuse = max((diffuseStrenght * dot(normal, lightDirection)), 0.0f) * lightColor;

    float spec = dot(viewDir, reflectDir);
    spec = max(spec, 0.0f);
    spec = powf(spec, shininess);
    float4 specular = specularStrength * spec * lightColor;

    float4 result = (ambient + diffuse + specular) * objectColor;

    return clamp(result, 0.0f, 1.0f);
}

__host__ __device__ float3 LumenOpus::PhongModel(
    const float4& mat, 
    const float3& rayOrigin, 
    const float3& spherePosition, 
    const float3& lightPosition, 
    const float3& lightColor, 
    const float3& hitPoint, 
    const float3& objectColor)
{
    float3 normal = normalize(hitPoint - spherePosition);
    float3 viewDir = normalize(rayOrigin - spherePosition);
    float3 lightDirection = normalize(lightPosition - hitPoint);
    float3 reflectDir = reflect(-lightDirection, normal);

    const float& ambientStrength = mat.x;
    const float& diffuseStrenght = mat.y;
    const float& specularStrength = mat.z;
    const float& shininess = mat.w;

    float3 ambient = ambientStrength * lightColor;
    float3 diffuse = max((diffuseStrenght * dot(normal, lightDirection)), 0.0f) * lightColor;

    float spec = dot(viewDir, reflectDir);
    spec = max(spec, 0.0f);
    spec = powf(spec, shininess);
    float3 specular = specularStrength * spec * lightColor;

    float3 result = (ambient + diffuse + specular) * objectColor;

    return clamp(result, 0.0f, 1.0f);
}

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

int main2()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}



// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
